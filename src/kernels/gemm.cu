#include "hip/hip_runtime.h"
#include "kernels/gemm.h"
#include "utils.h"
#include "constants.h"
#include <cstddef>


/**
    s1: [batch_size, rows, N]
    s2: [batch_size, cols, N]
    output: [batch_size, rows, cols]
*/
__global__ void gemm_transpose_kernel(
    const float* s1, const float* s2, float* output,
    size_t batch_size, size_t rows, size_t cols, size_t N) {

    __shared__ float s1_shared[TILE_SIZE][TILE_SIZE];
    __shared__ float s2_shared[TILE_SIZE][TILE_SIZE];
    const float* base_s1 = s1 + blockIdx.z * rows * N;
    const float* base_s2 = s2 + blockIdx.z * cols * N;
    float result = 0;
    /**
     * writing to [blockIdx.z, blockIdx.y * TILE_SIZE + threadIdx.y, blockIdx.x * TILE_SIZE + threadIdx.x]
     * using s1[blockIdx.z, blockIdx.y * TILE_SIZE : blockIdx.y * TILE_SIZE + TILE_SIZE, :]
     * using s2[blockIdx.z, blockIdx.x * TILE_SIZE : blockIdx.x * TILE_SIZE + TILE_SIZE, :]
     */
    for (size_t i = 0; i < N; i += TILE_SIZE) {
        if (blockIdx.y * TILE_SIZE + threadIdx.y < rows && i + threadIdx.x < N) {
            s1_shared[threadIdx.y][threadIdx.x] = base_s1[(blockIdx.y * TILE_SIZE + threadIdx.y) * N + i + threadIdx.x];
        } else {
            s1_shared[threadIdx.y][threadIdx.x] = 0.0f;
        }
        if (blockIdx.x * TILE_SIZE + threadIdx.y < cols && i + threadIdx.x < N) {
            // this is to ensure coalensce, always use threadIdx.x as the last dim
            s2_shared[threadIdx.y][threadIdx.x] = base_s2[(blockIdx.x * TILE_SIZE + threadIdx.y) * N + i + threadIdx.x];
        } else {
            s2_shared[threadIdx.y][threadIdx.x] = 0.0f;
        }
        __syncthreads();
        
        #pragma unroll
        for (size_t j = 0; j < TILE_SIZE; ++j) {
            result += (s1_shared[threadIdx.y][j] * s2_shared[threadIdx.x][j]);
        }
        __syncthreads();
    }

    if (blockIdx.y * TILE_SIZE + threadIdx.y < rows && blockIdx.x * TILE_SIZE + threadIdx.x < cols) {
        output[blockIdx.z * rows * cols + (blockIdx.y * TILE_SIZE + threadIdx.y) * cols + blockIdx.x * TILE_SIZE + threadIdx.x] = result;
    }
}

void launch_gemm_transpose_kernel(const float* s1, const float* s2, float* output,
    size_t batch_size, size_t rows, size_t cols, size_t N) {

    dim3 gridDim(ceil_div(cols, TILE_SIZE), ceil_div(rows, TILE_SIZE), batch_size);
    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    gemm_transpose_kernel<<<gridDim, blockDim>>>(s1, s2, output, batch_size, rows, cols, N);
    CUDA_CHECK_LAST();
}

/**
    s1: [batch_size, rows, N]
    s2: [batch_size, N, cols]
    output: [batch_size, rows, cols]
*/
__global__ void gemm_bias_kernel(
    const float* s1, Stride3D s1_stride,
    const float* s2, Stride3D s2_stride,
    const float* bias, Stride3D bias_stride,
    float* output, Stride3D output_stride,
    size_t batch_size, size_t rows, size_t N, size_t cols) {
    __shared__ float s1_shared[TILE_SIZE][TILE_SIZE];
    __shared__ float s2_shared[TILE_SIZE][TILE_SIZE];
    const float* base_s1 = s1 + (blockIdx.z * s1_stride.batch_stride);
    const float* base_s2 = s2 + (blockIdx.z * s2_stride.batch_stride);
    float default_bias = 0;
    if (bias == nullptr) {
        bias = &default_bias;
    }
    const float* base_bias = bias + (blockIdx.z * bias_stride.batch_stride);
    float result = 0;
    for (size_t i = 0; i < N; i += TILE_SIZE) {
        if ((blockIdx.y * TILE_SIZE + threadIdx.y) < rows && (i + threadIdx.x) < N) {
            s1_shared[threadIdx.y][threadIdx.x] = base_s1[(blockIdx.y * TILE_SIZE + threadIdx.y) * s1_stride.row_stride + (i + threadIdx.x) * s1_stride.col_stride];
        } else {
            s1_shared[threadIdx.y][threadIdx.x] = 0;
        }

        if ((blockIdx.x * TILE_SIZE + threadIdx.x) < cols && (i + threadIdx.y) < N) {
            s2_shared[threadIdx.y][threadIdx.x] = base_s2[(i + threadIdx.y) * s2_stride.row_stride + (blockIdx.x * TILE_SIZE + threadIdx.x) * s2_stride.col_stride];
        } else {
            s2_shared[threadIdx.y][threadIdx.x] = 0;
        }
        __syncthreads();

        #pragma unroll
        for (size_t j = 0; j < TILE_SIZE; ++j) {
            result += (s1_shared[threadIdx.y][j] * s2_shared[j][threadIdx.x]);
        }
        __syncthreads();
    }
    if (blockIdx.y * TILE_SIZE + threadIdx.y < rows && blockIdx.x * TILE_SIZE + threadIdx.x < cols) {
        output[blockIdx.z * output_stride.batch_stride
            + (blockIdx.y * TILE_SIZE + threadIdx.y) * output_stride.row_stride
            + (blockIdx.x * TILE_SIZE + threadIdx.x) * output_stride.col_stride]
            = result + base_bias[(blockIdx.y * TILE_SIZE + threadIdx.y) * bias_stride.row_stride
                + ((blockIdx.x * TILE_SIZE + threadIdx.x) * bias_stride.col_stride)];
    }
}


void launch_gemm_kernel(const float* s1, const float* s2, float* output, size_t batch_size, size_t rows, size_t N, size_t cols) {
    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((cols + TILE_SIZE - 1) / TILE_SIZE, (rows + TILE_SIZE - 1) / TILE_SIZE, batch_size);
    const float* null_float = nullptr;
    gemm_bias_kernel<<<gridDim, blockDim>>>(
        s1, Stride3D{rows * N, N, 1},
        s2, Stride3D{N * cols, cols, 1},
        null_float, Stride3D{0, 0, 0},
        output, Stride3D{rows * cols, cols, 1},
        batch_size, rows, N, cols);
    CUDA_CHECK_LAST();
}

void launch_gemm_bias_kernel(
    const float* s1, Stride3D s1_stride,
    const float* s2, Stride3D s2_stride,
    const float* bias, Stride3D bias_stride,
    float* output, Stride3D output_stride,
    size_t batch_size, size_t rows, size_t N, size_t cols) {
    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((cols + TILE_SIZE - 1) / TILE_SIZE, (rows + TILE_SIZE - 1) / TILE_SIZE, batch_size);
    gemm_bias_kernel<<<gridDim, blockDim>>>(
        s1, s1_stride,
        s2, s2_stride,
        bias, bias_stride,
        output, output_stride,
        batch_size, rows, N, cols);
    CUDA_CHECK_LAST();
}

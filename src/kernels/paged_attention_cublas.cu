#include "hip/hip_runtime.h"
#include "constants.h"
#include "tensor.hpp"
#include "utils.h"
#include "kernels/self_attention_inference_optimized.h"
#include <cassert>
#include <cstdlib>
#include <hipblas.h>
#include "kernels/paged_attention.h"
#include "kernels/gemm.h"

/**
 * page_table: [n_batch, n_sequence / PAGE_BLOCK_SIZE]
 * lengths: [n_batch]
 * latest_embs: [n_batch, embs]
 */
__global__ void get_latest_batch_embs(
    const float** page_table, const int* lengths,
    float* latest_embs,
    int n_batch, int n_sequence, int emb_dim) {

    int i_batch = blockIdx.y;
    int cur_length = lengths[i_batch];
    if (cur_length == 0) {
        return;
    }
    int i_sequence = cur_length - 1;
    int i_dim = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ const float* page_pos;
    if (threadIdx.x == 0) {
        page_pos = page_table[i_batch * (n_sequence / PAGE_BLOCK_SIZE) + i_sequence / PAGE_BLOCK_SIZE];
    }
    __syncthreads();
    if (i_dim < emb_dim) {
        latest_embs[i_batch * emb_dim + i_dim] = get_page_table_value(page_pos, i_batch, n_sequence, i_sequence, emb_dim, PAGE_BLOCK_SIZE, i_dim, INP_EMB_EMB_OFFSET);
    }
}

/**
 * page_table: [n_batch, n_sequence / PAGE_BLOCK_SIZE]
 * lengths: [n_batch]
 * latest_k, latest_v: [n_batch, embs]
 */
__global__ void save_to_page_table(
    float** page_table, const int* lengths, 
    const float* latest_k, const float* latest_v,
    int n_batch, int n_sequence, int emb_dim) {
    
    int i_batch = blockIdx.y;
    int cur_length = lengths[i_batch];
    if (cur_length == 0) {
        return;
    }
    int i_sequence = cur_length - 1;
    int i_dim = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float* page_pos;
    if (threadIdx.x == 0) {
        page_pos = page_table[i_batch * (n_sequence / PAGE_BLOCK_SIZE) + i_sequence / PAGE_BLOCK_SIZE];
    }
    __syncthreads();
    if (i_dim < emb_dim) {
        set_page_table_value(page_pos, i_batch, n_sequence, i_sequence, emb_dim, PAGE_BLOCK_SIZE, i_dim, K_CACHE_EMB_OFFSET, latest_k[i_batch * emb_dim + i_dim]);
        set_page_table_value(page_pos, i_batch, n_sequence, i_sequence, emb_dim, PAGE_BLOCK_SIZE, i_dim, V_CACHE_EMB_OFFSET, latest_v[i_batch * emb_dim + i_dim]);
    }
}

/**
 * page_table: [n_batch, n_sequence / PAGE_BLOCK_SIZE]
 * lengths: [n_batch]
 * latest_k, latest_v: [n_batch, embs]
 */
__global__ void assert_emb_equal(
    const float* q_to_check, const int* lengths, 
    const float* latest_emb,
    int n_batch, int n_sequence, int emb_dim) {
    
    int i_batch = blockIdx.y;
    int cur_length = lengths[i_batch];
    if (cur_length == 0) {
        return;
    }
    int i_sequence = cur_length - 1;
    int i_dim = blockIdx.x * blockDim.x + threadIdx.x;

    // __shared__ float* page_pos;
    // if (threadIdx.x == 0) {
    //     page_pos = page_table[i_batch * (n_sequence / PAGE_BLOCK_SIZE) + i_sequence / PAGE_BLOCK_SIZE];
    // }
    __syncthreads();
    if (i_dim < emb_dim) {
        assert(abs(latest_emb[i_batch * emb_dim + i_dim] - q_to_check[i_batch * emb_dim + i_dim]) < 0.001);
    }
}

/**
 * page_table: [n_batch, n_sequence / PAGE_BLOCK_SIZE]
 * lengths: [n_batch]
 * wq, wk, wv: [emb_dim, emb_dim]
 * q_output: [n_batch, emb_dim]
 * temp_placeholder: [n_batch, emb_dim]
 */
void launch_get_latest_k_q_v_paged_attention_cublas(
    TensorFloatPoint& page_table, const TensorInt& lengths,
    TensorFloat& latest_emb,
    const TensorFloat& wk, const TensorFloat& wq,
    const TensorFloat& wv, TensorFloat& q_output, TensorFloat& temp_placeholder,
    hipblasHandle_t& handle, int n_sequence, const TensorFloat& q_to_check) {

    int n_batch = page_table.shape()[0];
    int emb_dim = wq.shape()[0];
    
    dim3 gridDim(ceil_div(emb_dim, TILE_SIZE_SQUARE), n_batch);
    get_latest_batch_embs<<<gridDim, TILE_SIZE_SQUARE>>>((const float**)page_table.data(), lengths.data(), latest_emb.data(), n_batch, n_sequence, emb_dim);
    CUDA_CHECK_LAST();

    float alpha = 1.0f, beta = 0.0f;
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, emb_dim, n_batch, emb_dim, &alpha, wk.data(),
        emb_dim, latest_emb.data(), emb_dim, &beta, q_output.data(), emb_dim));
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, emb_dim, n_batch, emb_dim, &alpha, wv.data(),
        emb_dim, latest_emb.data(), emb_dim, &beta, temp_placeholder.data(), emb_dim));
    save_to_page_table<<<gridDim, TILE_SIZE_SQUARE>>>(page_table.data(), lengths.data(), q_output.data(), temp_placeholder.data(), n_batch, n_sequence, emb_dim);
    CUDA_CHECK_LAST();
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, emb_dim, n_batch, emb_dim, &alpha, wq.data(),
        emb_dim, latest_emb.data(), emb_dim, &beta, q_output.data(), emb_dim));
    // launch_gemm_kernel(latest_emb.data(), wq.data(), q_output.data(), 1, n_batch, emb_dim, emb_dim);
    assert_emb_equal<<<gridDim, TILE_SIZE_SQUARE>>>(q_output.data(), lengths.data(), q_to_check.data(), n_batch, n_sequence, emb_dim);
    CUDA_CHECK_LAST();
}


/**
 * - page_table: [n_batch, n_sequence / PAGE_BLOCK_SIZE], input_embedding + k_cache + v_cache
 *      Each pointer points to a memory block of (3 * embedding_dim * PAGE_BLOCK_SIZE)
 * - lengths: [n_batch], the token lengths for each batch
 * - wk, wq, wv: [emb_dim, emb_dim]: since we don't have the following feed-forward layer, 
 * input_dim should be equal to output_dim
 * - new_batch_idx: [n_batch], but only n_new_items are used
 * - q_output: [n_batch, emb_dim]
 * - qkt_output: [n_batch, n_sequence]
 * - attention_result: [n_batch, emb_dim]
 */
void paged_attention_with_cublas(
    TensorFloatPoint& page_table,
    const TensorInt& lengths,
    const TensorFloat& wk,
    const TensorFloat& wq,
    const TensorFloat& wv,
    const TensorInt& new_batch_idx,
    TensorFloat& q_output, TensorFloat& qkt_output, TensorFloat& attention_result,
    TensorFloat& latest_emb, TensorFloat& temp_placeholder,
    int n_new_items, int n_sequence, hipblasHandle_t& handle) {

    launch_fill_new_k_v_cache_paged_attention(page_table, new_batch_idx, lengths, wk, wv, n_new_items, n_sequence);

    // launch_get_latest_k_q_v_paged_attention_cublas(page_table, lengths, latest_emb, wk, wq, wv, q_output, temp_placeholder, handle, n_sequence);

    launch_qkt_paged_attention(q_output, page_table, lengths, qkt_output);

    launch_softmax_in_place_with_lengths(qkt_output, lengths);

    launch_softmax_v_paged_attention(qkt_output, page_table, attention_result, lengths);
}

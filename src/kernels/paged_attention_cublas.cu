#include "hip/hip_runtime.h"
#include "constants.h"
#include "tensor.hpp"
#include "utils.h"
#include "kernels/self_attention_inference_optimized.h"
#include <cassert>
#include <cstdlib>
#include <hipblas.h>
#include "kernels/paged_attention.h"
#include "kernels/templated_kernels.cuh"

/**
 * page_table: [n_batch, n_sequence / PAGE_BLOCK_SIZE]
 * lengths: [n_batch]
 * latest_embs: [n_batch, embs]
 */
__global__ void get_latest_batch_embs(
    const float** page_table, const int* lengths,
    float* latest_embs,
    int n_batch, int n_sequence, int emb_dim) {

    int i_batch = blockIdx.y;
    int cur_length = lengths[i_batch];
    if (cur_length == 0) {
        return;
    }
    int i_sequence = cur_length - 1;
    int i_dim = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ const float* page_pos;
    if (threadIdx.x == 0) {
        page_pos = page_table[i_batch * (n_sequence / PAGE_BLOCK_SIZE) + i_sequence / PAGE_BLOCK_SIZE];
    }
    __syncthreads();
    if (i_dim < emb_dim) {
        latest_embs[i_batch * emb_dim + i_dim] = get_page_table_value(page_pos, i_batch, n_sequence, i_sequence, emb_dim, PAGE_BLOCK_SIZE, i_dim, INP_EMB_EMB_OFFSET);
    }
}


/**
 * page_table: [n_batch, n_sequence / PAGE_BLOCK_SIZE]
 * lengths: [n_batch]
 * latest_k, latest_v: [n_batch, embs]
 */
__global__ void save_to_page_table(
    float** page_table, const int* lengths, 
    const float* latest_k, const float* latest_v,
    int n_batch, int n_sequence, int emb_dim) {
    
    int i_batch = blockIdx.y;
    int cur_length = lengths[i_batch];
    if (cur_length == 0) {
        return;
    }
    int i_sequence = cur_length - 1;
    int i_dim = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float* page_pos;
    if (threadIdx.x == 0) {
        page_pos = page_table[i_batch * (n_sequence / PAGE_BLOCK_SIZE) + i_sequence / PAGE_BLOCK_SIZE];
    }
    __syncthreads();
    if (i_dim < emb_dim) {
        set_page_table_value(page_pos, i_batch, n_sequence, i_sequence, emb_dim, PAGE_BLOCK_SIZE, i_dim, K_CACHE_EMB_OFFSET, latest_k[i_batch * emb_dim + i_dim]);
        set_page_table_value(page_pos, i_batch, n_sequence, i_sequence, emb_dim, PAGE_BLOCK_SIZE, i_dim, V_CACHE_EMB_OFFSET, latest_v[i_batch * emb_dim + i_dim]);
    }
}

/**
 * page_table: [n_batch, n_sequence / PAGE_BLOCK_SIZE]
 * lengths: [n_batch]
 * wq, wk, wv: [emb_dim, emb_dim]
 * q_output: [n_batch, emb_dim]
 * temp_placeholder: [n_batch, emb_dim]
 */
void launch_get_latest_k_q_v_paged_attention_cublas(
    TensorFloatPoint& page_table, const TensorInt& lengths,
    TensorFloat& latest_emb,
    const TensorFloat& wk, const TensorFloat& wq,
    const TensorFloat& wv, TensorFloat& q_output, TensorFloat& temp_placeholder,
    hipblasHandle_t& handle, int n_sequence) {

    int n_batch = page_table.shape()[0];
    int emb_dim = wq.shape()[0];
    
    dim3 gridDim(ceil_div(emb_dim, TILE_SIZE_SQUARE), n_batch);
    get_latest_batch_embs<<<gridDim, TILE_SIZE_SQUARE>>>((const float**)page_table.data(), lengths.data(), latest_emb.data(), n_batch, n_sequence, emb_dim);
    CUDA_CHECK_LAST();

    float alpha = 1.0f, beta = 0.0f;
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, emb_dim, n_batch, emb_dim, &alpha, wk.data(),
        emb_dim, latest_emb.data(), emb_dim, &beta, q_output.data(), emb_dim));
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, emb_dim, n_batch, emb_dim, &alpha, wv.data(),
        emb_dim, latest_emb.data(), emb_dim, &beta, temp_placeholder.data(), emb_dim));
    save_to_page_table<<<gridDim, TILE_SIZE_SQUARE>>>(page_table.data(), lengths.data(), q_output.data(), temp_placeholder.data(), n_batch, n_sequence, emb_dim);
    CUDA_CHECK_LAST();
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, emb_dim, n_batch, emb_dim, &alpha, wq.data(),
        emb_dim, latest_emb.data(), emb_dim, &beta, q_output.data(), emb_dim));
}

constexpr int WARPSIZE = 32;

/**
 * page_table: [n_batch, n_sequence / PAGE_BLOCK_SIZE]
 * new_batch_idx: [n_new_batch]
 * lengths: [n_batch]
 * wk: [emb_dim, emb_dim]
 * wv: [emb_dim, emb_dim]
 *
 * Try to use warp tiling optimization
 */
template <const int BM, const int BN, const int BK, const int WM, const int WN,
          const int TM, const int TN, const int WNITER, const int N_THREADS>
__global__ void fill_new_k_v_cache_paged_attention_warp_tiling(
    float** page_table, const int* new_batch_idx,
    const int* lengths,
    const float* wk, const float* wv,
    int n_sequence, int emb_dim) {

    int batch_idx = new_batch_idx[blockIdx.z];
    int cur_length = lengths[batch_idx];

    // place the warp in the block
    int warpIdx = threadIdx.x / WARPSIZE;
    int warpRow = warpIdx / (BN / WN);
    int warpCol = warpIdx % (BN / WN);

    // The starting position is [output_block_row_id * BM, output_block_col_id * BN]
    int cRow = blockIdx.y;
    int cCol = blockIdx.x;

    constexpr int WMITER = (WM * WN) / (WARPSIZE * TM * TN * WNITER);
    constexpr int WSUBM = WM / WMITER;
    constexpr int WSUBN = WN / WNITER;

    static_assert(BM % TM == 0, "BM must be divisible by TM");
    static_assert(BN % TN == 0, "BN must be divisible by TN");
    static_assert(N_THREADS * 4 % BK == 0, "N_THREADS * 4 must be divisible by BK");
    static_assert(N_THREADS * 4 % BN == 0, "N_THREADS * 4 must be divisible by BK");
    static_assert(WSUBN % TN == 0, "WSUBN must be divisible by TN");
    static_assert(WSUBM % TM == 0, "WSUBM must be divisible by TM");
    static_assert(WSUBM <= WARP_SIZE, "WSUBM must be less than or equal to WARP_SIZE to avoid bank conflicts");
    static_assert(WSUBN <= WARP_SIZE, "WSUBN must be less than or equal to WARP_SIZE to avoid bank conflicts");
    static_assert(TN >= 4, "TN must be greater than or equal to 4 to ensure we can use float4");
    
    __shared__ float shared_inp[BM * BK];
    __shared__ float shared_wk[BK * BN];
    __shared__ float shared_wv[BK * BN];

    if (cRow * BM >= cur_length) {
        return;
    }
    page_table += batch_idx * (n_sequence / PAGE_BLOCK_SIZE);

    constexpr int rowStrideA = (N_THREADS * 4) / BK;
    constexpr int rowStrideB = (N_THREADS * 4) / BN;

    // Place the thread in the block
    int innerRowA = (threadIdx.x / (BK / 4));
    int innerColA = (threadIdx.x % (BK / 4));
    int innerRowB = (threadIdx.x / (BN / 4));
    int innerColB = (threadIdx.x % (BN / 4));

    float reg_inp[TM * WMITER] = {0};
    float reg_wk[TN * WNITER] = {0};
    float reg_wv[TN * WNITER] = {0};
    float thread_result_k[TM * WMITER * TN * WNITER] = {0};
    float thread_result_v[TM * WMITER * TN * WNITER] = {0};


    // place the thread in the warp
    int threadIdxInWarp = threadIdx.x % WARPSIZE;
    int threadRowInWarp = threadIdxInWarp / (WSUBN / TN);
    int threadColInWarp = threadIdxInWarp % (WSUBN / TN);

    for (int bk_idx = 0; bk_idx < emb_dim; bk_idx += BK) {
        load_from_page_table<N_THREADS, rowStrideA, BM>(
            page_table, n_sequence, emb_dim,
            shared_inp, innerRowA, innerColA, cRow * BM, bk_idx, cur_length);
        load_from_kv<N_THREADS, rowStrideB, BN, BK>(
            wk, emb_dim,
            shared_wk, innerRowB, innerColB, bk_idx, cCol * BN);
        load_from_kv<N_THREADS, rowStrideB, BN, BK>(
            wv, emb_dim,
            shared_wv, innerRowB, innerColB, bk_idx, cCol * BN);
        __syncthreads();

        process_result<WM, WN, WMITER, WNITER, TM, TN, BM, BN, BK, WSUBM, WSUBN>(
            shared_inp, shared_wk, shared_wv,
            reg_inp, reg_wk, reg_wv,
            thread_result_k, thread_result_v,
            warpRow, warpCol, threadRowInWarp, threadColInWarp);
        __syncthreads();
    }

    int cached_paged_idx = -1;
    float* cached_page_pos = nullptr;
    for (int wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx) {
        for (int resIdxM = 0; resIdxM < TM; ++resIdxM) {
            int output_row = cRow * BM + warpRow * WM + wSubRowIdx * WSUBM + threadRowInWarp * TM + resIdxM;
            if (output_row < cur_length) {
                int page_idx = output_row / PAGE_BLOCK_SIZE;
                if (cached_paged_idx != page_idx) {
                    cached_paged_idx = page_idx;
                    cached_page_pos = page_table[page_idx];
                }

                for (int wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx) {
                    for (int resIdxN = 0; resIdxN < TN; resIdxN += 4) {
                        int output_col = cCol * BN + warpCol * WN + wSubColIdx * WSUBN + threadColInWarp * TN + resIdxN;
                        if (output_col < emb_dim) {
                            assert(output_col + 3 < emb_dim);
                            int k_offset = (output_row % PAGE_BLOCK_SIZE) * emb_dim * 3 + emb_dim * K_CACHE_EMB_OFFSET + output_col;
                            int v_offset = (output_row % PAGE_BLOCK_SIZE) * emb_dim * 3 + emb_dim * V_CACHE_EMB_OFFSET + output_col;
                            reinterpret_cast<float4*>(cached_page_pos + k_offset)[0] = reinterpret_cast<float4*>(thread_result_k + (wSubRowIdx * TM + resIdxM) * WNITER * TN + wSubColIdx * TN + resIdxN)[0];
                            reinterpret_cast<float4*>(cached_page_pos + v_offset)[0] = reinterpret_cast<float4*>(thread_result_v + (wSubRowIdx * TM + resIdxM) * WNITER * TN + wSubColIdx * TN + resIdxN)[0];
                        }
                    }
                }
            }
        }
    }
}

void launch_fill_new_k_v_cache_paged_attention_warp_tiling(
    TensorFloatPoint page_table, const TensorInt& new_batch_idx, const TensorInt& lengths,
    const TensorFloat& wk, const TensorFloat& wv, int n_new_items, int n_sequence) {

    if (n_new_items == 0) {
        return;
    }

    int n_batch = page_table.shape()[0];
    assert(page_table.shape()[1] == n_sequence / PAGE_BLOCK_SIZE && n_sequence % PAGE_BLOCK_SIZE == 0);
    int emb_dim = wk.shape()[0];
    assert(wk.shape()[0] == wk.shape()[1]);
    constexpr int N_THREADS = 128;
    constexpr int BM = 64, BN = 64, BK = 64, WM = 32, WN = 32, TM = 4, TN = 4, WNITER = 2;

    fill_new_k_v_cache_paged_attention_warp_tiling<BM, BN, BK, WM, WN, TM, TN, WNITER, N_THREADS>
        <<<dim3(ceil_div(emb_dim, BN), ceil_div(n_sequence, BM), n_new_items), N_THREADS>>>(
            page_table.data(), new_batch_idx.data(), lengths.data(),
            wk.data(), wv.data(), n_sequence, emb_dim);
    
    CUDA_CHECK_LAST();
}


/**
 * - page_table: [n_batch, n_sequence / PAGE_BLOCK_SIZE], input_embedding + k_cache + v_cache
 *      Each pointer points to a memory block of (3 * embedding_dim * PAGE_BLOCK_SIZE)
 * - lengths: [n_batch], the token lengths for each batch
 * - wk, wq, wv: [emb_dim, emb_dim]: since we don't have the following feed-forward layer, 
 * input_dim should be equal to output_dim
 * - new_batch_idx: [n_batch], but only n_new_items are used
 * - q_output: [n_batch, emb_dim]
 * - qkt_output: [n_batch, n_sequence]
 * - attention_result: [n_batch, emb_dim]
 */
void paged_attention_with_cublas(
    TensorFloatPoint& page_table,
    const TensorInt& lengths,
    const TensorFloat& wk,
    const TensorFloat& wq,
    const TensorFloat& wv,
    const TensorInt& new_batch_idx,
    TensorFloat& q_output, TensorFloat& qkt_output, TensorFloat& attention_result,
    TensorFloat& latest_emb, TensorFloat& temp_placeholder,
    int n_new_items, int n_sequence, hipblasHandle_t& handle) {

    launch_fill_new_k_v_cache_paged_attention(page_table, new_batch_idx, lengths, wk, wv, n_new_items, n_sequence);

    launch_get_latest_k_q_v_paged_attention_cublas(page_table, lengths, latest_emb, wk, wq, wv, q_output, temp_placeholder, handle, n_sequence);

    launch_qkt_paged_attention(q_output, page_table, lengths, qkt_output);

    launch_softmax_in_place_with_lengths(qkt_output, lengths);

    launch_softmax_v_paged_attention(qkt_output, page_table, attention_result, lengths);
}
